
#include <hip/hip_runtime.h>
#include <stdio.h>

// these are just for timing measurments
#include <time.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int DSIZE = 4096;
const int block_size = 16;
const float A_val = 1.0f;
const float B_val = 2.0f;

// matrix multiply (naive) kernel: C = A * B
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  int idy = threadIdx.y+blockDim.y*blockIdx.y;

  if ((idx < ds) && (idy < ds)){
    float temp = 0;
    for (int i = 0; i < ds; i++)
      temp += A[idy * ds + i] * B[idy * ds + i];
    C[idy*ds+idx] = temp;
  }
}

int main(){

  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

  clock_t t0, t1, t2;
  double t1sum=0.0;
  double t2sum=0.0;

  t0 = clock();

  h_A = new float[DSIZE*DSIZE];
  h_B = new float[DSIZE*DSIZE];
  h_C = new float[DSIZE*DSIZE];
  for (int i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = A_val;
    h_B[i] = B_val;
    h_C[i] = 0;}

  t1 = clock();
  t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
  printf("Init took %f seconds.  Begin compute\n", t1sum);

  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  dim3 block(block_size, block_size);
  dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);
  mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

  t2 = clock();
  t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
  printf ("Done. Compute took %f seconds\n", t2sum);

  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  for (int i = 0; i < DSIZE*DSIZE; i++) if (h_C[i] != A_val*B_val*DSIZE) {printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], A_val*B_val*DSIZE); return -1;}
  printf("Success!\n"); 

  return 0;
}
  
